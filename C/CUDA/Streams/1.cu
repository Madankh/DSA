#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)

template <typename>

template<typename T>
void check(T err, const char* const func, const char* const file, const int line){
    if(err != hipSuccess){
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s)\%s\" \n", file, line, static_cast<unsigned int>(err), hipGetErrorString(err), func);
        exit(EXIT_FAILURE);
    }
}

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements){
        C[i] = A[i] + B[i];
    }
}

int main(){
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    hipStream_t  stream1, stream2;

    //Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // # Initialize host array
    for(int i=0; i<numElements; i++){
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    };

    // Allocate device memory
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_A, size));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_B, size));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_C, size));
    return 0;
}