#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hip/hip_fp16.h>

#define M 3
#define K 4
#define N 2

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} 

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error in %s:%d: %d\n", __FILE__, __LINE__, status); \
        exit(EXIT_FAILURE); \
    } \
}

#undef PRINT_MATRIX
#define PRINT_MATRIX(mat, rows, cols) \
    for (int i = 0; i < rows; i++) { \
        for (int j = 0; j < cols; j++) \
            printf("%8.3f ", mat[i * cols + j]); \
        printf("\n"); \
    } \
    printf("\n");

void cpu_matmul(float *A, float *B, float *C){
    for
}