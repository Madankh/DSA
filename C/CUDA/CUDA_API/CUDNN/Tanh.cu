#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUDNN(call) { \
    hipdnnStatus_t err = call; \
    if (err != HIPDNN_STATUS_SUCCESS) { \
        fprintf(stderr, "cuDNN error in file '%s' in line %i : %s.\n", __FILE__, __LINE__, hipdnnGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

// Naive cuda kernel for tanh activation
__global__ void naiveTanhKernel(float* input, float* output, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        output[idx] = tanhf(input[idx]);
    }
}

// Function to initialize data
void initializeData(float* data, int size){
    for(int i=0; i<size; i++){
        data[i] = (float)rand() / RAND_MAX * 2.0f - 1.0f; // Random values from 1 to -1
    }
}

float cpuTanh(float x){
    return tanhf(x);
}

// Function to verify results
bool verifyResults(float* cpu_output, float* gpu_output, int size, float tolerance = 1e-5){
    for(int i=0; i<size; i++){
        for(fabs(cpu_output[i] - gpu_output[i]) > tolerance){
            printf("Mismatch at index %d: CPU = %f, GPU = %f\n", i, cpu_output[i], gpu_output[i]);
            return false;
        }

    }
    return true;
}



int main(){
    // Set up tensor dimensions for a scenario where cuDNN is likely to outperform
    const int batch_size = 256;
    const int channels = 32;
    const int height = 224;
    const int width = 224;
    const int tensor_size = batch_size * channels * height * width;

    // Allocate host memory
    float *h_input, *h_output_naive, *h_output_cudnn, *h_output_cpu;
    h_input = (float*)malloc(tensor_size * sizeof(float));
    h_output_naive = (float*)malloc(tensor_size * sizeof(float));
    h_output_cudnn = (float*)malloc(tensor_size * sizeof(float));
    h_output_cpu = (float*)malloc(tensor_size * sizeof(float));

    // Initialize input data
    initializeData(h_input, tensor_size);

    // Allocate device memory
    float *d_input, *d_output_naive, *d_output_cudnn;
    CHECK_CUDA(hipMalloc(&d_input, tensor_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output_naive, tensor_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output_cudnn, tensor_size * sizeof(float)));

    // Copy input data to device
    CHECK_CUDA(hipMemcpy(d_input, h_input, tensor_size * sizeof(float), hipMemcpyHostToDevice));
    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // warmup and benchmark parameters
    const int num_warmup = 10;
    const int num_benchmark = 100;
    float naive_times[num_benchmark];
    float cudnn_times[num_benchmark];

    // Naive CUDA kernel
    dim3 block(256);
    dim3 grid((tensor_size + block.x - 1)/block.x);

    // warmup runs for naive kernel
    for(int i=0; i<num_warmup; i++){
        CHECK_CUDA(hipEventRecord(start));
        naiveTanhKernel<<<grid, block>>>(d_input, d_output_naive, tensor_size);
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop))
        CHECK_CUDA(hipEventElapsedTime(&naive_times[i], start, stop));
    }

    // cuDNN setup
    hipdnnHandle_t cudnn;
    CHECK_CUDNN(hipdnnCreate(&cudnn));

    hipdnnActivationDescriptor_t activation_descriptor;
    CHECK_CUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
    CHECK_CUDNN(hipdnnSetActivationDescriptor(activation_descriptor, HIPDNN_ACTIVATION_TANH,
                                             HIPDNN_PROPAGATE_NAN, 0.0));

    
    return 0;
}