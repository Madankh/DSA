#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error in %s:%d: %d\n", __FILE__, __LINE__, status); \
        exit(EXIT_FAILURE); \
    } \
}
// Naive cuda kernel for tanh activation
__global__ void naiveTanhKernel(float* input, float* output, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        output[idx] = tanhf(input[idx]);
    }
}

// Function to initialize data
void initializeData(float* data, int size){
    for(int i=0; i<size; i++){
        data[i] = (float)rand() / RAND_MAX * 2.0f - 1.0f; // Random values from 1 to -1
    }
}

float cpuTanh(float x){
    return tanhf(x);
}

// Function to verify results
bool verifyResults(float* cpu_output, float* gpu_output, int size, float tolerance = 1e-5){
    for(int i=0; i<size; i++){
        for(fabs(cpu_output[i] - gpu_output[i]) > tolerance){
            printf("Mismatch at index %d: CPU = %f, GPU = %f\n", i, cpu_output[i], gpu_output[i]);
            return false;
        }

    }
    return true;
}



int main(){
    // Set up tensor dimensions for a scenario where cuDNN is likely to outperform
    const int batch_size = 256;
    const int channels = 32;
    const int height = 224;
    const int width = 224;
    const int tensor_size = batch_size * channels * height * width;

    // Allocate host memory
    float *h_input, *h_output_naive, *h_output_cudnn, *h_output_cpu;
    h_input = (float*)malloc(tensor_size * sizeof(float));
    h_output_naive = (float*)malloc(tensor_size * sizeof(float));
    h_output_cudnn = (float*)malloc(tensor_size * sizeof(float));
    h_output_cpu = (float*)malloc(tensor_size * sizeof(float));

    // Initialize input data
    initializeData(h_input, tensor_size);

    // Allocate device memory
    float *d_input, *d_output_naive, *d_output_cudnn;
    CHECK_CUDA(hipMalloc(*d_input, tensor_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(*d_output_naive, tensor_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(*d_output_cudnn, tensor_size * sizeof(float)));
    
    return 0;
}