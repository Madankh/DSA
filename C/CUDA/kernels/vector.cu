#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <time.h>
#include<hip/hip_runtime.h>
#include <windows.h>
#define N 10000000
#define BLOCK_SIZE 256

// CPU vector addition
void vector_add_cpu(float *a, float *b, float *c, int n){
    for(int i=0; i < n; i++){
        c[i] = a[i] + b[i];
    }
}

// CUDA kernel for vector addition
__global__ void vector_add_gpu(float* a, float* b, float* c , int n){
    int i = blockIdx.x * blockDim.x * threadIdx.x;
    if (i<n){
        c[i] = a[i] + b[i];
    }
}

// Initialize vector with random values
void init_vector(float *vec, int n){
    for(int i=0; i<n; i++){
        vec[i] = (float)rand() / RAND_MAX;
    }
}

// Function to measure execution time
double get_time(){
    LARGE_INTEGER freq, counter;
    QueryPerformanceFrequency(&freq);  // Get the frequency of the high-resolution timer
    QueryPerformanceCounter(&counter); // Get the current value of the counter
    return (double)counter.QuadPart / freq.QuadPart;
}

int main(){
    float *h_a, *h_b, *h_c_cpu, *h_c_gpu;
    float *d_a, *d_b, *d_c;
    size_t size = N * sizeof(float);

    // Allocate host memory;
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);

    h_c_cpu = (float*)malloc(size);
    h_c_gpu = (float*)malloc(size);

    // Initialize vectors
    srand(time(NULL));
    init_vector(h_a, N);
    init_vector(h_b, N);

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b , size);
    hipMalloc(&d_c , size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    

    return 0;
}
