#include "hip/hip_runtime.h"
<<<<<<< HEAD
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <windows.h>
#include <math.h>

=======
#include<stdio.h>
#include<stdlib.h>
#include <time.h>
#include<hip/hip_runtime.h>
#include <windows.h>
>>>>>>> 22bf1c6a42b3f9ef1f242423df72d0b85cad005f
#define N 10000000
#define BLOCK_SIZE 256

// CPU vector addition
<<<<<<< HEAD
void vector_add_cpu(float *a, float *b, float *c, int n) {
    for (int i = 0; i < n; i++) {
=======
void vector_add_cpu(float *a, float *b, float *c, int n){
    for(int i=0; i < n; i++){
>>>>>>> 22bf1c6a42b3f9ef1f242423df72d0b85cad005f
        c[i] = a[i] + b[i];
    }
}

// CUDA kernel for vector addition
<<<<<<< HEAD
__global__ void vector_add_gpu(float* a, float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
=======
__global__ void vector_add_gpu(float* a, float* b, float* c , int n){
    int i = blockIdx.x * blockDim.x * threadIdx.x;
    if (i<n){
>>>>>>> 22bf1c6a42b3f9ef1f242423df72d0b85cad005f
        c[i] = a[i] + b[i];
    }
}

// Initialize vector with random values
<<<<<<< HEAD
void init_vector(float *vec, int n) {
    for (int i = 0; i < n; i++) {
=======
void init_vector(float *vec, int n){
    for(int i=0; i<n; i++){
>>>>>>> 22bf1c6a42b3f9ef1f242423df72d0b85cad005f
        vec[i] = (float)rand() / RAND_MAX;
    }
}

// Function to measure execution time
<<<<<<< HEAD
double get_time() {
    LARGE_INTEGER freq, counter;
    QueryPerformanceFrequency(&freq);
    QueryPerformanceCounter(&counter);
    return (double)(counter.QuadPart * 1000) / freq.QuadPart;
}

int main() {
=======
double get_time(){
    LARGE_INTEGER freq, counter;
    QueryPerformanceFrequency(&freq);  // Get the frequency of the high-resolution timer
    QueryPerformanceCounter(&counter); // Get the current value of the counter
    return (double)counter.QuadPart / freq.QuadPart;
}

int main(){
>>>>>>> 22bf1c6a42b3f9ef1f242423df72d0b85cad005f
    float *h_a, *h_b, *h_c_cpu, *h_c_gpu;
    float *d_a, *d_b, *d_c;
    size_t size = N * sizeof(float);

<<<<<<< HEAD
    // Allocate host memory
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
=======
    // Allocate host memory;
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);

>>>>>>> 22bf1c6a42b3f9ef1f242423df72d0b85cad005f
    h_c_cpu = (float*)malloc(size);
    h_c_gpu = (float*)malloc(size);

    // Initialize vectors
<<<<<<< HEAD
    srand((unsigned int)time(NULL));
=======
    srand(time(NULL));
>>>>>>> 22bf1c6a42b3f9ef1f242423df72d0b85cad005f
    init_vector(h_a, N);
    init_vector(h_b, N);

    // Allocate device memory
    hipMalloc(&d_a, size);
<<<<<<< HEAD
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Warm up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        vector_add_cpu(h_a, h_b, h_c_cpu, N);
        vector_add_gpu<<<num_blocks, BLOCK_SIZE>>>(d_a, d_b, d_c, N);
        hipDeviceSynchronize();
    }

    // Benchmark CPU implementation
    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        vector_add_cpu(h_a, h_b, h_c_cpu, N);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;

    // Benchmark GPU implementation
    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        vector_add_gpu<<<num_blocks, BLOCK_SIZE>>>(d_a, d_b, d_c, N);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;

    // Print results
    printf("CPU average time: %f milliseconds\n", cpu_avg_time);
    printf("GPU average time: %f milliseconds\n", gpu_avg_time);
    printf("Speedup: %f\n", cpu_avg_time / gpu_avg_time);

    // Verify results
    hipMemcpy(h_c_gpu, d_c, size, hipMemcpyDeviceToHost);
    bool correct = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c_cpu[i] - h_c_gpu[i]) > 1e-5) {
            correct = false;
            break;
        }
    }
    printf("Results are %s\n", correct ? "correct" : "incorrect");

    // Free memory
    free(h_a);
    free(h_b);
    free(h_c_cpu);
    free(h_c_gpu);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
=======
    hipMalloc(&d_b , size);
    hipMalloc(&d_c , size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    

    return 0;
}
>>>>>>> 22bf1c6a42b3f9ef1f242423df72d0b85cad005f
