#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>
#include<Windows.h>
#define M 256  // Number of rows in A and C
#define K 512   //  Number of columns in A and rows in B
#define N 256 // Number of columns in B and C
#define BLOCK_SIZE 32

/// Example 3x2 @ 2x4 = 3x4 --? (M X K) @ (K x N) = (M X N)
// A = [[1,2], 
//      [3,4],
//      [5,6]]

// B = [[7,8,9,10],
//     [11, 12 , 13 , 14]]

// C = [[29, 32 , 35, 38],
//     [65, 72, 79, 86],
//     [101, 112, 123, 134]]


// CPU matrix multitplication
void matmul_cpu(float *A, float *B, float *C, int m, int k, int n){
    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            float sum = 0.0f;
            for(int l = 0; l<k; l++){
                sum+= A[i * k + l] * B[l * n +  j];
            }
            C[i * n + j] = sum;
        }
    }
}

/// Example 3x2 @ 2x4 = 3x4 --? (M X K) @ (K x N) = (M X N)

// CUDA kernel for matrix multiplication
__global__ void matmul_gpu(float *A, float *B, float *C, int m, int k, int n){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m && col < n){
        float sum = 0.0;
        for(int l = 0; l<k; l++){
            sum += A[row * k + l] * B[l * n + col];
        }
        C[row * n + col] = sum;
    }
}

void init_matrix(float *mat, int rows, int cols){
    for(int i=0; i<rows*cols; i++){
        mat[i] = (float)rand()/RAND_MAX;
    }
}

// Function to measure execution time
double get_time(){
    LARGE_INTEGER freq, counter;
    QueryPerformanceFrequency(&freq);  // Get the frequency of the high-resolution timer
    QueryPerformanceCounter(&counter); // Get the current value of the counter
    return (double)counter.QuadPart / freq.QuadPart;
}

int main(){
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_B, *d_C;
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    // Allocate host memory
    h_A = (float*)(malloc)(size_A);
    h_B = (float*)(malloc)(size_B);
    h_C_cpu = (float*)(malloc)(size_C);
    h_C_gpu = (float*)malloc(size_C);
    // Allocate device memory

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1)/BLOCK_SIZE, (M  + BLOCK_SIZE - 1)/ BLOCK_SIZE);
    printf("Performing warm-up runs....\n");
    for(int i=0; i<3; i++){
        matmul_cpu(h_A, h_B, h_C_cpu, M,K,N);
        matmul_gpu<<<gridDim, blockDim>>>(d_A, d_B,d_C, M, K , N);
        hipDeviceSynchronize();
    }

        // Benchmark CPU implementation
    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;

    // Benchmark GPU implementation
    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        matmul_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;

    printf("CPU average time: %f microseconds\n", (cpu_avg_time * 1e6f));
    printf("GPU average time: %f microseconds\n", (gpu_avg_time * 1e6f));
    printf("Speedup: %fx\n", cpu_avg_time / gpu_avg_time);

    bool correct = true;
    for (int i = 0; i < M * N; i++) {
        if (fabs(h_C_cpu[i] - h_C_gpu[i]) > 1e-5) {
            correct = false;
            break;
        }
    }
    printf("Results are %s\n", correct ? "correct" : "incorrect");


    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    

    return 0;
}