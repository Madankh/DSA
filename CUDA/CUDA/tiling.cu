
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<assert.h>
#include<math.h>

// static shmem calculation for convenience (Int 16 x 16 matrix )
# define SHHEM_SIZE 16 * 16 * 4

__global__ void tiledMatrixMul(int *a, int *b, int *c, int n, int tile_size){
    // Two statically-sized pieces of shared memory
    __shared__ int A[SHHEM_SIZE];
    __shared__ int B[SHHEM_SIZE];

    // shorten these parameters for clean re-use
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // calculate global row and column positions for this thread
    int row = by * tile_size  + ty;
    int col = bx * tile_size + tx;
    // Intermediate sum for element being written
    // sweep tiles over entire matrix
    for(int i=0;i<(n/tile_size); i++){
        /*
        */
       A[(ty * tile_size) + tx] = a[row * n + (i * tile_size + tx)];
       B[(ty * tile_size) + ty] = b[(i * tile_size * n + ty + n) + col];
       // Ensure all threads have loaded their data before processding
       __syncthreads();

       // calculate all temp values fopr this tile
    }

}